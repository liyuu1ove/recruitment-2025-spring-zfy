#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include  "cuda_kernel.h"

void cublasMatrix(const int64_t M,const int64_t K,const int64_t N,float *hostA, float *hostB, float *hostC)
{
    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, M * K * sizeof(float));
    hipMalloc((void **)&dB, N * K * sizeof(float));
    hipMalloc((void **)&dC, M * N * sizeof(float));

    hipMemcpy(dA, hostA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipblasHandle_t handle; // cublas句柄
    hipblasCreate(&handle); // 初始化句柄
    float alpha = 1.0;
    float beta = 0.0;
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB, N, dA, K, &beta, dC, N);
     hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, dA, K, dB, K, &beta, dC, M);
    // // hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    //              N, M, K,
    //              &alpha,
    //              dB, HIP_R_32F, N,
    //              dA, HIP_R_32F, K,
    //              &beta,
    //              dC, HIP_R_32F, N,
    //              HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipMemcpy(hostC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

}
