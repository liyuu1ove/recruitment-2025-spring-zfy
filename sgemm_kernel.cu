#include <hipblas.h>
#include <hip/hip_runtime.h>


void cublasMatrix(const int64_t M,const int64_t K,const int64_t N,float *hostA, float *hostB, float *hostC)
{
    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, M * K * sizeof(float));
    hipMalloc((void **)&dB, N * K * sizeof(float));
    hipMalloc((void **)&dC, M * N * sizeof(float));

    hipMemcpy(dA, hostA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipblasHandle_t handle; // cublas句柄
    hipblasCreate(&handle); // 初始化句柄
    float alpha = 1.0;
    float beta = 0.0;
    // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, M, K, &alpha, dB, N, dA, K, &beta, dC, N);
     hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, dA, K, dB, K, &beta, dC, M);
    // // cublasGemmEx(handle, CUBLAS_OP_N, CUBLAS_OP_N,
    //              N, M, K,
    //              &alpha,
    //              dB, CUDA_R_32F, N,
    //              dA, CUDA_R_32F, K,
    //              &beta,
    //              dC, CUDA_R_32F, N,
    //              CUDA_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipMemcpy(hostC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

}

void sgemm_cublas(const int64_t M, const int64_t N, const int64_t K, float *A, float *B, float *C) {
    float alpha = 1.0f, beta = 0.0f;

    typedef float(*A_tensor_t)[K];
    typedef float(*B_tensor_t)[K];
    typedef float(*C_tensor_t)[M];
    A_tensor_t A_tensor = (A_tensor_t)A;
    B_tensor_t B_tensor = (B_tensor_t)B;
    C_tensor_t C_tensor = (C_tensor_t)C;

    // 初始化cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // 将数据从主机复制到设备
    hipblasSetMatrix(M, K, sizeof(float), A_tensor, K, d_A, K);
    hipblasSetMatrix(N, K, sizeof(float), B_tensor, K, d_B, K);
    hipblasSetMatrix(N, M, sizeof(float), C_tensor, M, d_C, M);

    // 执行SGEMM，注意B和C已经被转置
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, K, &alpha, d_A, M, d_B, N, &beta, d_C, M);

    // 将结果从设备复制回主机
    //cublasGetMatrix(M, N, sizeof(float), d_C, M, C_tensor, M);
    hipMemcpy(C_tensor,d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 销毁cuBLAS句柄
    hipblasDestroy(handle);
}
